#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "Timer.h"
#include "Kernel/Reduce.cuh"
#include "Kernel/Scan.cuh"
#include "Kernel/RadixSort.cuh"
#include "Test/InsertionSortTest.cuh"

#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

__device__
int add(int a, int b) {
    return a + b;
}

int serialReduce(int* ints, int size) {
    int sum = 0;
    for (int i = 0; i < size; ++i) {
        sum += ints[i];
    }
    return sum;
}

void reducePlayground() {
    printf("Begin reducePlayground\n");

    constexpr int kSize = 1024 * 1024 * 32;

    int* in = (int*) malloc(kSize *sizeof(int));

    for (int i = 0; i < kSize; ++i) {
        in[i] = 1;
    }

    int* d_in;
    int* d_out;

    hipMalloc(&d_in, kSize * sizeof(int));
    hipMalloc(&d_out, kSize * sizeof(int));

    hipMemcpy(d_in, in, kSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_out, 0, kSize * sizeof(int));

    {
        Timer time;
        int result = Reduce::reduce<int, add>(d_in, d_out, kSize);
        printf("\nGPU result: %d\n", result);
    }

    {
        Timer time;
        int result = serialReduce(in, kSize);
        printf("\nSER result: %d\n", result);
    }

    free(in);

    hipFree(d_in);
    hipFree(d_out);

    printf("\nEnd reducePlayground\n\n");
}

void checkScanErrors(int* input, int * output, int* d_out, int size) {
    hipMemcpy(output, d_out, size * sizeof(int), hipMemcpyDeviceToHost);

    int expected = 0;
    int numErrors = 0;
    for (int i = 0; i < size; ++i) {
        expected += input[i];

        int actual = output[i];
        if (expected != actual) {
            //printf("Mismatch i %d exp %d act %d\n", i, expected, actual);
            numErrors += 1;
        }
    }

    if (numErrors > 0) {
        printf("Num scan errors %d\n", numErrors);
    }
}

void scanPlayground() {
    printf("Begin scanPlayground\n");

    constexpr int kSize = 1024 * 1024 * 4;

    int* input = (int*) malloc(kSize * sizeof(int));
    int* output = (int*) malloc(kSize * sizeof(int));

    for (int i = 0; i < kSize; ++i) {
        input[i] = 1;
    }

    int* d_in;
    int* d_out;
    int* d_offsets;

    checkCudaErrors(hipMalloc(&d_in, kSize * sizeof(int)));
    hipMalloc(&d_out, kSize * sizeof(int));
    hipMalloc(&d_offsets, kSize * sizeof(int));

    hipMemcpy(d_in, input, kSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_out, 0, kSize * sizeof(int));
    hipMemset(d_offsets, 0, kSize * sizeof(int));

    {
        Timer timer;
        for (int i = 0; i < 100; ++i) {
            Scan::scan<int, add>(d_in, d_out, d_offsets, kSize);
            checkScanErrors(input, output, d_out, kSize);
        }
    }

    free(input);
    free(output);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_offsets);

    printf("\nEnd scanPlayground\n\n");
}

void radixSortPlayground() {

    printf("Begin radixSortPlayground\n");

    constexpr int kSize = 1024 * 1024;
    
    unsigned int* input = (unsigned int*) malloc(kSize * sizeof(unsigned int));
    unsigned int* output = (unsigned int*) malloc(kSize * sizeof(unsigned int));

    for (int i = 0; i < kSize; ++i) {
        input[i] = i % 100;
        output[i] = 0;
    }
    
    unsigned int* d_a;
    unsigned int* d_b;
    uint4* d_flags_a;
    uint4* d_flags_b;

    hipMalloc(&d_a, kSize * sizeof(unsigned int));
    hipMalloc(&d_b, kSize * sizeof(unsigned int));
    hipMalloc(&d_flags_a, kSize * sizeof(uint4));
    hipMalloc(&d_flags_b, kSize * sizeof(uint4));

    hipMemcpy(d_a, input, kSize * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemset(d_b, 0, kSize * sizeof(unsigned int));

    unsigned int* sorted;
    {
        Timer timer;
        sorted = RadixSort::sort<unsigned int>(d_a, d_b, d_flags_a, d_flags_b, kSize);
    }

    hipMemcpy(output, sorted, kSize * sizeof(unsigned int), hipMemcpyDeviceToHost);

    free(input);
    free(output);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_flags_a);
    hipFree(d_flags_b);

    printf("\nEnd radixSortPlayground\n");
}

// For some mysterious reason, reduce and scan are non deterministic and suffer from errors when threadsPerBlock is not 1024

int main() {
    reducePlayground();
    scanPlayground();
    InsertionSortTest::run();
    //radixSortPlayground();
}
