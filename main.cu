#include <stdio.h>
#include <iostream>
#include <chrono>
#include "Timer.h"
#include "Reduce.cuh"
#include "Scan.cuh"

__device__
int add(int a, int b) {
    return a + b;
}

int serialReduce(int* ints, int size) {
    int sum = 0;
    for (int i = 0; i < size; ++i) {
        sum += ints[i];
    }
    return sum;
}

void reducePlayground() {
    printf("Begin reducePlayground\n");

    constexpr int kSize = 1024 * 1024 * 128;

    int* in = (int*) malloc(kSize *sizeof(int));

    for (int i = 0; i < kSize; ++i) {
        in[i] = 1;
    }

    int* d_in;
    int* d_out;

    hipMalloc(&d_in, kSize * sizeof(int));
    hipMalloc(&d_out, kSize * sizeof(int));

    hipMemcpy(d_in, in, kSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_out, 0, kSize * sizeof(int));

    {
        Timer time;
        int result = Reduce::reduce<int, add>(d_in, d_out, kSize);
        printf("\nGPU result: %d\n", result);
    }

    {
        Timer time;
        int result = serialReduce(in, kSize);
        printf("\nSER result: %d\n", result);
    }

    free(in);

    hipFree(d_in);
    hipFree(d_out);

    printf("\nEnd reducePlayground\n\n");
}

void scanPlayground() {
    printf("Begin scanPlayground\n");

    constexpr int kSize = 1024;

    int* input = (int*) malloc(kSize * sizeof(int));
    int* output = (int*) malloc(kSize * sizeof(int));

    for (int i = 0; i < kSize; ++i) {
        input[i] = 1;
    }

    int* d_a;
    int* d_b;

    hipMalloc(&d_a, kSize * sizeof(int));
    hipMalloc(&d_b, kSize * sizeof(int));

    hipMemcpy(d_a, input, kSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_b, 0, kSize * sizeof(int));

    {
        Timer timer;
        Scan::scan<int, add>(d_a, d_b,kSize);
    }

    hipMemcpy(output, d_b, kSize * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < kSize; ++i) {
        //printf("%d %d\n", i, output[i]);
    }

    printf("\nEnd scanPlayground\n\n");
}

int main() {
    reducePlayground();
    scanPlayground();
}
