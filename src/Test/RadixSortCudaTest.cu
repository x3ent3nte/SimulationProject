#include "hip/hip_runtime.h"
#include <Test/RadixSortCudaTest.cuh>

#include <Cuda/RadixSort.cuh>

#include <Utils/Timer.h>

std::vector<uint32_t> RadixSortCudaTest::run(const std::vector<uint32_t>& numbers) {

    unsigned int* d_a;
    unsigned int* d_b;
    uint4* d_flags;

    const size_t memorySize = numbers.size() * sizeof(uint32_t);

    hipMalloc(&d_a, memorySize);
    hipMalloc(&d_b, memorySize);
    hipMalloc(&d_flags, numbers.size() * sizeof(uint4) * 2);

    hipMemcpy(d_a, numbers.data(), memorySize, hipMemcpyHostToDevice);
    hipMemset(d_b, 0, memorySize);
    hipMemset(d_flags, 0, numbers.size() * sizeof(uint4) * 2);

    uint32_t* d_sorted;
    {
        Timer timer("Radix Sort Cuda");
        d_sorted = RadixSort::sort<unsigned int>(d_a, d_b, d_flags, numbers.size());
    }

    std::vector<uint32_t> sorted(numbers.size());
    hipMemcpy(sorted.data(), d_sorted, memorySize, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_flags);

    return sorted;
}
