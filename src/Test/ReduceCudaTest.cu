#include <Test/ReduceCudaTest.cuh>

#include <Cuda/Reduce.cuh>
#include <Utils/Timer.h>

namespace {

    __device__
    Collision earliestCollision(Collision a, Collision b) {
        if (a.time < b.time) {
            return a;
        } else {
            return b;
        }
    }
} // end namespace anonymous

Collision ReduceCudaTest::run(const std::vector<Collision>& data) {

    size_t bufferSize = data.size() * sizeof(Collision);

    Collision* d_collisionsOne;
    Collision* d_collisionsTwo;
    hipMalloc(&d_collisionsOne, bufferSize);
    hipMalloc(&d_collisionsTwo, bufferSize);

    hipMemcpy(d_collisionsOne, data.data(), bufferSize, hipMemcpyHostToDevice);

    Collision result;
    {
        Timer timer("Reduce CUDA");
        result = Reduce::reduce<Collision, earliestCollision>(d_collisionsOne, d_collisionsTwo, data.size());
    }

    hipFree(d_collisionsOne);
    hipFree(d_collisionsTwo);

    return result;
}
