#include <Test/ReduceCudaTest.cuh>

#include <Cuda/Reduce.cuh>
#include <Utils/Timer.h>

namespace {

    __device__
    ReducerUtil::Collision earliestCollision(ReducerUtil::Collision a, ReducerUtil::Collision b) {
        if (a.time < b.time) {
            return a;
        } else {
            return b;
        }
    }
} // end namespace anonymous

ReducerUtil::Collision ReduceCudaTest::run(const std::vector<ReducerUtil::Collision>& data) {

    size_t bufferSize = data.size() * sizeof(ReducerUtil::Collision);

    ReducerUtil::Collision* d_collisionsOne;
    ReducerUtil::Collision* d_collisionsTwo;
    hipMalloc(&d_collisionsOne, bufferSize);
    hipMalloc(&d_collisionsTwo, bufferSize);

    hipMemcpy(d_collisionsOne, data.data(), bufferSize, hipMemcpyHostToDevice);

    ReducerUtil::Collision result;
    {
        Timer timer("Reduce CUDA");
        result = Reduce::reduce<ReducerUtil::Collision, earliestCollision>(d_collisionsOne, d_collisionsTwo, data.size());
    }

    hipFree(d_collisionsOne);
    hipFree(d_collisionsTwo);

    return result;
}
