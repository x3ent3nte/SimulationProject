#include <Test/InsertionSortCudaTest.cuh>

#include <Kernel/InsertionSort.cuh>
#include <Utils/Timer.h>

#include <stdio.h>
#include <string>
#include <iostream>
#include <vector>

namespace {

__device__
int floatGreater(float a, float b) {
    return a > b;
}

} // namespace anonymous

std::vector<float> InsertionSortCudaTest::run(const std::vector<float>& data) {

    std::vector<float> dataCopy(data);

    size_t size = dataCopy.size() * sizeof(float);

    float* d_data;
    int* d_needsSorting;
    hipMalloc(&d_data, size);
    hipMalloc(&d_needsSorting, sizeof(int));

    hipMemcpy(d_data, dataCopy.data(), size, hipMemcpyHostToDevice);

    InsertionSort::sort<float, floatGreater>(d_data, d_needsSorting, size);

    hipMemcpy(dataCopy.data(), d_data, size, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_needsSorting);

    return dataCopy;
}
