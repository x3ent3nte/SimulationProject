#include <Test/ScanCudaTest.cuh>

#include <Cuda/Scan.cuh>
#include <Utils/Timer.h>

namespace {
    __device__
    int add(int a, int b) {
        return a + b;
    }
} // namespace anonymous

std::vector<int> ScanCudaTest::run(const std::vector<int>& data) {
    size_t bufferSize = data.size() * sizeof(int);

    int* d_in;
    int* d_out;
    int* d_offsets;

    hipMalloc(&d_in, bufferSize);
    hipMalloc(&d_out, bufferSize);
    hipMalloc(&d_offsets, bufferSize);

    hipMemcpy(d_in, data.data(), bufferSize, hipMemcpyHostToDevice);
    hipMemset(d_out, 0, bufferSize);
    hipMemset(d_offsets, 0, bufferSize);

    {
        Timer timer("Scan CUDA");
        Scan::scan<int, add>(d_in, d_out, d_offsets, data.size());
    }

    std::vector<int> result(data.size());
    hipMemcpy(result.data(), d_out, bufferSize, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_offsets);

    return result;
}
