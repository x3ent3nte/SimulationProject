
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 1024

__global__
void vectorAdd(int* a, int* b, int* c, int size) {
    int i = (blockIdx.x * SIZE) + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    
    int* a = (int*) malloc(SIZE * SIZE * sizeof(int));
    int* b = (int*) malloc(SIZE * SIZE * sizeof(int));
    int* c = (int*) malloc(SIZE * SIZE * sizeof(int));

    int* d_a;
    int* d_b;
    int* d_c;

    hipMalloc(&d_a, SIZE * SIZE * sizeof(int));
    hipMalloc(&d_b, SIZE * SIZE * sizeof(int));
    hipMalloc(&d_c, SIZE * SIZE * sizeof(int));

    for (int i = 0; i < SIZE * SIZE; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<SIZE, SIZE>>>(d_a, d_b, d_c, SIZE * SIZE);

    hipMemcpy(c, d_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; ++i) {
        std::cout << c[i] << "\n";
    }

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
